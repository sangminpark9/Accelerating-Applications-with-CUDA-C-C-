
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

__global__ void helloGPU()
{
  printf("Hello also from the CPU.\n");
}

int main()
{

  helloCPU();
  helloGPU<<<1,1>>>();

  hipDeviceSynchronize();
}
