
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void loop(int N)
{ 
  printf("This is iteration number %d\n", threadIdx.x);
}

int main()
{

  int N = 10;
  loop<<<1,10>>>(N);
  hipDeviceSynchronize();
}
