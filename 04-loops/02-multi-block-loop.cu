
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void loop(int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("This is iteration number %d\n", i);
}

int main()
{
  int N = 10;
  loop<<<5,2>>>(N);
  hipDeviceSynchronize();
}
